extern "C" {

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <omp.h>

}

const int device0 = 0;
#define BLOCK_SIZE 16

void __global__ jacobi_gpu2(int N, double delta, int kMAX, double *f, double *u_new, double *u_old) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    double scalar = 1.0/4;
    if (i <= N && j <= N && i > 0 && j > 0) {
        u_new[i*N + j] = scalar * (u_old[(i-1)*N + j] + u_old[(i+1)*N + j] + u_old[i*N + (j-1)] + u_old[i*N + (j+1)] +delta*f[i*N + j]);
    }
}

int main(int argc, char *argv[]) {

    // warm up:
    double *dummy_d;
    hipSetDevice(device0);
    hipMalloc((void**)&dummy_d, 0);

    int kMAX, N,i,j;

    if (argc == 3) {
        N = atoi(argv[1]);
        kMAX = atoi(argv[2]);
    }
    else {
        // use default N
        N = 200;
        kMAX = 5000;
    }
    double delta = (2.0 / N) * (2.0 / N);

    // allocate mem
    double *h_f, *h_u_new, *h_u_old, *d_f, *d_u_new, *d_u_old;

    int size_f = N * N * sizeof(double);
    int size_u_new = N * N * sizeof(double);
    int size_u_old = N * N * sizeof(double);

    //Allocate memory on device
    hipSetDevice(device0);
    hipMalloc((void**)&d_f, size_f);
    hipMalloc((void**)&d_u_new, size_u_new);
    hipMalloc((void**)&d_u_old, size_u_old);
    //Allocate memory on host
    hipHostMalloc((void**)&h_f, size_f, hipHostMallocDefault);
    hipHostMalloc((void**)&h_u_new, size_u_new, hipHostMallocDefault);
    hipHostMalloc((void**)&h_u_old, size_u_old, hipHostMallocDefault);

    // initilize boarder
    for (i = 0; i <N; i++){
        for (j = 0; j < N; j++){
            if (i >= N * 0.5  &&  i <= N * 2.0/3.0  &&  j >= N * 1.0/6.0  &&  j <= N * 1.0/3.0)
                h_f[i*N + j] = 200.0;
            else
                h_f[i*N + j] = 0.0;

            if (i == (N - 1) || i == 0 || j == (N - 1)){
                h_u_new[i*N + j] = 20.0;
                h_u_old[i*N + j] = 20.0;
            }
            else{
                h_u_new[i*N + j] = 0.0;
                h_u_old[i*N + j] = 0.0;
            }
        }
    }

    //Copy memory CPU -> GPU
    double time_tmp = omp_get_wtime();
    hipMemcpy(d_f, h_f, size_f, hipMemcpyHostToDevice);
    hipMemcpy(d_u_new, h_u_new, size_u_old, hipMemcpyHostToDevice);
    hipMemcpy(d_u_old, h_u_old, size_u_old, hipMemcpyHostToDevice);
    double time_IO_1 = omp_get_wtime() - time_tmp;

    // do program
    int k = 0;
    dim3 dim_grid(((N+BLOCK_SIZE-1) / BLOCK_SIZE), ((N+BLOCK_SIZE-1) / BLOCK_SIZE));
    dim3 dim_block(BLOCK_SIZE, BLOCK_SIZE);
    double *temp, time_compute = omp_get_wtime();
    while (k < kMAX) {
        // Set u_old = u
        temp = d_u_new;
        d_u_new = d_u_old;
        d_u_old = temp;
        jacobi_gpu2<<<dim_grid,dim_block>>>(N, delta, kMAX, d_f, d_u_new, d_u_old);
        hipDeviceSynchronize();
        k++;
    }/* end while */
    double tot_time_compute = omp_get_wtime() - time_compute;
    // end program

    //Copy memory GPU -> CPU
    time_tmp = omp_get_wtime();
    hipMemcpy(h_u_new, d_u_new, size_u_new, hipMemcpyDeviceToHost);
    double time_IO_2 = omp_get_wtime() - time_tmp;

    tot_time_compute += time_IO_1 + time_IO_2;

    // stats
    double GB = 1.0e-09;
    double flop = kMAX * (double)(N) * (double)(N) * 10.0;
    double gflops  = (flop / tot_time_compute) * GB;
    double memory  = size_f + size_u_new + size_u_old;
    double memoryGBs  = memory * GB * (1 / tot_time_compute);

    printf("%d\t", N);
    printf("%g\t", memory); // footprint
    printf("%g\t", gflops); // Gflops
    //printf("%g\t", memoryGBs); // bandwidth GB/s
    printf("%g\t", tot_time_compute); // total time
    //printf("%g\t", time_IO_1 + time_IO_2); // I/O time
    //printf("%g\t", tot_time_compute); // compute time
    printf("# GPU2\n");

    // free allocated mem
    hipFree(d_f), hipFree(d_u_new), hipFree(d_u_old);
    hipHostFree(h_f), hipHostFree(h_u_new), hipHostFree(h_u_old);
    // end program
    return(0);
}